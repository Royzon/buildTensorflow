#include "hip/hip_runtime.h"
#include<vector>
#include<iostream>
#include <chrono> // for measuring performance
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

auto cpuVectorAddition(std::vector<int> &A, std::vector<int> &B) {
    auto start = std::chrono::high_resolution_clock::now();
    for(int i = 0;i< A.size();i++) {
        A[i] += B[i];
    }
    auto stop = std::chrono::high_resolution_clock::now(); 
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start); 
    std::cout <<"Speed of CPU vector Addition: " << duration.count() <<" micro seconds"<<std::endl; 
    return duration.count();
}

__global__ void add(int *a, int *b, int*c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

// GPU vector Addition using Pointers
auto gpuVectorAddition(std::vector<int> &A, std::vector<int> &B) {
    size_t n= A.size();

    int* h_A = A.data();
    int* h_B = B.data();

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, sizeof(int)*n);
    hipMalloc((void**)&d_b, sizeof(int)*n);
    hipMalloc((void**)&d_c, sizeof(int)*n);

    hipMemcpy((void *)d_a, h_A, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy((void *)d_b, h_B, sizeof(int)*n, hipMemcpyHostToDevice);

    // Timing stuff, record how many seconds it takes for this operation
    hipEvent_t launch_begin, launch_end;
    hipEventCreate(&launch_begin);
    hipEventCreate(&launch_end);

    // Warmup
    add<<<n,1>>>(d_a, d_b, d_c);// num blocks, num_threads
    float total_time = 0;
    // Get average of 100 runs
    for(int i = 0;i<100;i++) {
        hipEventRecord(launch_begin,0);
        add<<<n,1>>>(d_a, d_b, d_c);
        hipEventRecord(launch_end,0);
        hipEventSynchronize(launch_end);

        float time = 0;
        hipEventElapsedTime(&time, launch_begin, launch_end);
        total_time += time;
    }

    total_time /= 100;
    std::cout <<"Speed of GPU vector Addition: " << total_time <<" micro seconds"<<std::endl; 

    // Copy memory back and free stuff
    hipMemcpy((void *)h_A, d_c, sizeof(int)*n, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return total_time;
}

void vectorAdditionTest() {
    size_t n = 10000000;
    // std::cout<<n<<std::endl;

    std::vector<int> A(n,1);
    std::vector<int> B(n,-1);
    auto timeCpu = cpuVectorAddition(A,B);
    auto timeGpu = gpuVectorAddition(A,B);

    std::cout<<"Speedup over CPU is: "<< (float)timeCpu/timeGpu <<std::endl;
}

// Observation for CPU vs GPU compute in vector addition, the answer why is as follows:

// 1. CUDA has a start-up overhead. For "small" problems like this one, the startup overhead will outweigh any gains from using the GPU. 

int main() {

    // std::vector<int> A = {1,2,3};
    // std::vector<int> B = {3,4,5};

    // vectorAddition(A,B); // puts result in A
    // for(auto i: A) {
    //     std::cout<<i<<std::endl;
    // }

    vectorAdditionTest();

    return 1;
    
}